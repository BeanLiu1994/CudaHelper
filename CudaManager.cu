#include "CudaManager.h"

#ifdef UseCuda

#include <hip/hip_runtime.h>

#if __CUDACC_VER_MAJOR__ == 8
#include "common/helper_cuda_80.h"
#elif __CUDACC_VER_MAJOR__ == 9
#include "common/hip/hip_runtime_api.h"
#else
#define DEVICE_RESET
#endif
#include <stdexcept>
#include <cstdlib>
#include <cstdio>

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		DEVICE_RESET
		if (abort) throw std::runtime_error(hipGetErrorString(code));
	}
}
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

void CuPtr::CuMallocAndCopy()
{
	if (state == CuState::MallocFinisied)
		CuFree();
	gpuErrchk(hipMalloc((void**)&d_Ptr, Size));
	gpuErrchk(hipMemcpy(d_Ptr, Ptr, Size, hipMemcpyHostToDevice));
	state = CuState::MallocFinisied;
}
void CuPtr::CuMallocAndSetVal(int val)
{
	if (state == CuState::MallocFinisied)
		CuFree();
	gpuErrchk(hipMalloc((void**)&d_Ptr, Size));
	gpuErrchk(hipMemset(d_Ptr, val, Size));
	state = CuState::MallocFinisied;
}
void CuPtr::CuGetResult(void* OutPtr)
{
	if (OutPtr == nullptr)
		OutPtr = Ptr;
	//gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(OutPtr, d_Ptr, Size, hipMemcpyDeviceToHost));
}
void CuPtr::CuFree()
{
	if (state == CuState::MallocFinisied)
	{
		gpuErrchk(hipFree(d_Ptr));
		state = CuState::Deleted;
	}
}
void CuPtr::CuSyncDevice()
{
	gpuErrchk(hipDeviceSynchronize());
}




void CuPtr_Const::CuMallocAndCopy()
{
	if (state == CuState::MallocFinisied)
		CuFree();
	gpuErrchk(hipMalloc((void**)&d_Ptr, Size));
	gpuErrchk(hipMemcpy(d_Ptr, Ptr, Size, hipMemcpyHostToDevice));
	state = CuState::MallocFinisied;
}
void CuPtr_Const::CuMallocAndSetVal(int val)
{
	if (state == CuState::MallocFinisied)
		CuFree();
	gpuErrchk(hipMalloc((void**)&d_Ptr, Size));
	gpuErrchk(hipMemset(d_Ptr, val, Size));
	state = CuState::MallocFinisied;
}
void CuPtr_Const::CuFree()
{
	if (state == CuState::MallocFinisied)
	{
		gpuErrchk(hipFree(d_Ptr));
		state = CuState::Deleted;
	}
}
void CuPtr_Const::CuSyncDevice()
{
	gpuErrchk(hipDeviceSynchronize());
}

cudaInitializer cudaInitializer::item = cudaInitializer();
cudaInitializer::~cudaInitializer()
{
	if (cudaInitializer::CudaOK())
	{
		gpuErrchk(hipDeviceReset());
	}
}

int cudaInitializer::dev = -1;
cudaInitializer::cudaInitializer()
{

}
int cudaInitializer::Init()
{
	if (!cudaInitializer::CudaOK())
	{
		try
		{
			dev = findCudaDevice(0, nullptr);
		}
		catch (std::exception& e)
		{
			printf(("[Cuda Initial Failed] " + std::string(e.what()) + " .\n").c_str());
			return dev;
		}
		hipDeviceProp_t deviceProp;
		gpuErrchk(hipGetDeviceProperties(&deviceProp, dev));
		//printf("[Cuda Initial Succeed] GPU Device %d: \"%s\" with compute capability %d.%d\n", dev, deviceProp.name, deviceProp.major, deviceProp.minor);
	}
	return dev;
}

bool cudaInitializer::CudaOK()
{
	return dev != -1;
}
#endif