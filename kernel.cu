#include "hip/hip_runtime.h"
#include "kernel.h"
#include "CudaManager.h"
#include <vector>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// 测试1 常见用法
void test1(int *c, const int *a, const int *b, unsigned int size)
{
	CuPtr<const int> da(a, size);
	CuPtr<const int> db(b, size);
	CuPtr<int> dc(c, size);

    	addKernel<<<1, size>>>(
		dc.GetDevicePtr(),
		da.GetDevicePtr(), 
		db.GetDevicePtr()
		);
    
	dc.CuGetResult();
}

// 测试2 直接分配gpu空间,之后复制到c里
void test2(int *c, const int *a, const int *b, unsigned int size)
{
	CuPtr<const int> da(a, size);
	CuPtr<const int> db(b, size);
	CuPtr<int> dc(nullptr, size*sizeof(int));

	addKernel << <1, size >> >(
		dc.GetDevicePtr(),
		da.GetDevicePtr(), 
		db.GetDevicePtr()
		);

	dc.CuGetResult(c);
}

// 测试3 使用其他库分出的空间
void test3(std::vector<int>& c, const int *a, const int *b, unsigned int size)
{
	// 运行过程中不要使c重分空间
	CuPtr<const int> da(a, size);
	CuPtr<const int> db(b, size);
	CuPtr<int> dc(&(c[0]), size);

	addKernel << <1, size >> >(
		dc.GetDevicePtr(),
		da.GetDevicePtr(), 
		db.GetDevicePtr()
		);

	dc.CuGetResult();
}
