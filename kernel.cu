#include "hip/hip_runtime.h"
#include "kernel.h"
#include "CudaManager.h"
#include <vector>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// 测试1 常见用法
void test1(int *c, const int *a, const int *b, unsigned int size)
{
	CuPtr_Const da(a, size);
	CuPtr_Const db(b, size);
	CuPtr dc(c, size);

    addKernel<<<1, size>>>(
		(int*)dc.GetDevicePtr(),
		(int*)da.GetDevicePtr(), 
		(int*)db.GetDevicePtr()
		);
    
	dc.CuGetResult();
}

// 测试2 直接分配gpu空间,之后复制到c里
void test2(int *c, const int *a, const int *b, unsigned int size)
{
	CuPtr_Const da(a, size);
	CuPtr_Const db(b, size);
	CuPtr dc(nullptr, size*sizeof(int));

	addKernel << <1, size >> >(
		(int*)dc.GetDevicePtr(),
		(int*)da.GetDevicePtr(),
		(int*)db.GetDevicePtr()
		);

	dc.CuGetResult(c);
}

// 测试3 使用其他库分出的空间
void test3(std::vector<int>& c, const int *a, const int *b, unsigned int size)
{
	// 运行过程中不要使c重分空间
	CuPtr_Const da(a, size);
	CuPtr_Const db(b, size);
	CuPtr dc(&(c[0]), size);

	addKernel << <1, size >> >(
		(int*)dc.GetDevicePtr(),
		(int*)da.GetDevicePtr(),
		(int*)db.GetDevicePtr()
		);

	dc.CuGetResult();
}